#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <time.h> 
#include<stdlib.h>
#include<string.h>
#include <iostream>

#include "cudaHelperHeader.cuh"

__global__ void carpCuda(unsigned char* resimBolgesi, float* filtre, unsigned char* donecekBolge, int boy, int en) {
	int b = threadIdx.x;
	int e = threadIdx.y;
	int x = (int)*(resimBolgesi + b * en + e);
	float y = *(filtre + b * en + e);
	float z = (float)x * y;
	*(donecekBolge + b * en + e) = z;
}

int carpTopla(unsigned char* resimBolgesi, float* filtre, int boy, int en) {

	unsigned char* donecekBolge = (unsigned char*)malloc(sizeof(unsigned char) * boy * en);

	unsigned char* gpu_rb = (unsigned char*)malloc(sizeof(unsigned char) * boy * en);
	float* gpu_f = (float*)malloc(sizeof(float) * boy * en);
	unsigned char* gpu_db = (unsigned char*)malloc(sizeof(unsigned char) * boy * en);

	hipMalloc(&gpu_rb, sizeof(unsigned char) * boy * en);
	hipMalloc(&gpu_f, sizeof(float) * boy * en);
	hipMalloc(&gpu_db, sizeof(unsigned char) * boy * en);

	hipMemcpy(gpu_rb, resimBolgesi, boy * en * sizeof(unsigned char), hipMemcpyHostToDevice);
	hipMemcpy(gpu_f, filtre, boy * en * sizeof(float), hipMemcpyHostToDevice);

	int numBlocks = 1;
	dim3 threadsPerBlock(boy, en);
	carpCuda << < numBlocks, threadsPerBlock >> > (gpu_rb, gpu_f, gpu_db, boy, en);

	hipMemcpy(donecekBolge, gpu_db, boy * en * sizeof(unsigned char), hipMemcpyDeviceToHost);

	hipFree(gpu_rb);
	hipFree(gpu_f);
	hipFree(gpu_db);

	int toplam = 0;
	for (int i = 0; i < boy; i++) {
		for (int j = 0; j < en; j++) {
			toplam += (int)*(donecekBolge + i * en + j);
		}
	}

	return toplam;
}