#include <stdio.h>
#include <time.h> 
#include<stdlib.h>
#include<string.h>
#include <iostream>

#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc.hpp>

#include"convolutionLibHeader.h"
//#include "cudaHelperHeader.cuh"

#include "hip/hip_runtime.h"
#include ""

void renklidenemepaddingyokk() {
    float f[] = { 0.1,0.1,0.1,0.1,0.1,0.1,0.1,0.1,0.1 };
    Mat resim = imread("C:\\Users\\user\\source\\repos\\opencv_deneme1\\1812.jpg");//deneme1.bmp	
    unsigned char* r = mat2MatrisRenkli(resim);
    int kanal = 3;
    int filtreBoyutu[] = { 3, 3 };
    int resimBoyutu[] = { resim.rows, resim.cols, kanal };
    int kaydirma[] = { 1, 1 };
    unsigned char* r2 = konvolusyonRenkli(r, f, resimBoyutu, filtreBoyutu, kaydirma, false);//true	
    Mat resim3 = matris2MatRenkli(r2, resim.rows - 2, resim.cols - 2);
    matBas(new Mat[2]{ resim, resim3 }, 2);
}

void renklidenemepaddingvarr() {
    float f[] = { 0.1,0.1,0.1,0.1,0.1,0.1,0.1,0.1,0.1 };
    Mat resim = imread("C:\\Users\\user\\source\\repos\\opencv_deneme1\\1812.jpg");//deneme1.bmp	
    unsigned char* r = mat2MatrisRenkli(resim);
    int kanal = 3;
    int filtreBoyutu[] = { 3, 3 };
    int resimBoyutu[] = { resim.rows, resim.cols, kanal };
    int kaydirma[] = { 1, 1 };
    unsigned char* r2 = konvolusyonRenkli(r, f, resimBoyutu, filtreBoyutu, kaydirma, true);//true	
    Mat resim3 = matris2MatRenkli(r2, resim.rows, resim.cols);
    matBas(new Mat[2]{ resim, resim3 }, 2);
}

int main() {
    renklidenemepaddingvarr();
    return 0;
}
